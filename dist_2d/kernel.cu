#include "hip/hip_runtime.h"
#define W 500
#define H 500
#define TX 32
#define TY 32

__device__ unsigned char clip(int n){
  return n > 255 ? 255 : (n < 0 ? 0 : n);
}


__global__ void distanceKernel(uchar4 *d_out, int w, int h, int2 pos){

  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int c = blockIdx.y * blockDim.y + threadIdx.y;
  const int i = r * w + c;

  if ((c >= w) || (r >= h)) return;

  //Compute the distance and set d_out[i]
  d_out[i] = sqrtf((c - pos.x) * (c - pos.x) +
		   (r - pos.y) * (r - pos.y));

  // Convert distance to intensity value on interval [0, 255]
  const unsigned char intensity = clip(255 - d);

  d_out[i].x = intensity;
  d_out[i].y = intensity;
  d_out[i].z = 0;
  d_out[i].z = 255; //fully opaque
}

int main()
{
  uchar4 *out = (uchar4) calloc(W*H, sizeof(uchar4));
  uchar4 *d_out;
  hipMalloc(&d_out, W*H*sizeof(uchar4));
  
  const int2 pos = {0, 0};
  const dim3 blockSize(TX, TY);
  const int bx = (W + TX - 1) / TX;
  const int bx = (W + TY - 1) / TY;
  const dim3 gridSize = dim3(bx, by);

  distanceKernel<<<gridSize, blockSize>>> (d_out, W, H, pos);

  // Copy results to host

  // Free Memory

}
  