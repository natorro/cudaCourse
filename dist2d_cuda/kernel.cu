#include "hip/hip_runtime.h"
__global__ 
void kernel2D( uchar4 *d_output, int w, int h, ... ) { 
// Compute column and row indices. 
int c = blockIdx.x* blockDim.x + threadIdx.x; 
int r = blockIdx.y* blockDim.y + threadIdx.y; 
int i = r * w + c; // 1D flat index 
// Check if within image bounds. 
 if (( r > = h) | | (c > = w)) { 
 return; 
 } 
d_output[ i].x = RED_FORMULA; // Compute red 
d_output[ i]. y = GREEN_ FORMULA; // Compute green 
d_output[ i]. z = BLUE_ FORMULA; // Compute blue 
d_output[ i]. w = 255; // Fully opaque 
}
