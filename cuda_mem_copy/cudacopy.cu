
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
  int N;
  int *a, *b, *c;
  N = 10;		
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  
  hipMalloc(&c, N*sizeof(int));
  
  
  for (int i = 0; i < N; i++){
    a[i] = i;
  }
  
  hipMemcpy(c, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, c, N*sizeof(int), hipMemcpyDeviceToHost);
  
  for (int i = 0; i < N; i++){
    printf("%d ,", a[i]);
  }
  
  for (int i = 0; i < N; i++){
    printf("%d ,", b[i]);
  }
}
