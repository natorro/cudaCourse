#include "aux_functions.h"
#include <stdlib.h> // Supports dynamic memory management.
#include <stdio.h> // Supports printing

#define N 6144 // A large array size.
#define TPB 1024

int main()
{
  float *in = (float *)calloc(N, sizeof(float));
  float *d_in, *d_out;
  hipMalloc(&d_in, N*sizeof(float));
  hipMalloc(&d_out, N*sizeof(float));
  
  const float ref = 0.5f;

  for (int i = 0; i < N; ++i)
  {
    in[i] = scale(i, N);
  }
  
  hipMemcpy(d_in, in, N*sizeof(float), hipMemcpyHostToDevice);
  
  distanceArray<<<N/TPB, TPB>>>(d_out, d_in, ref, N);
  
  hipMemcpy(in, d_out, N*sizeof(float), hipMemcpyDeviceToHost);
 for(int i = 0; i < N; i++){
   printf("i = %2d: dist from %f to value is %f.\n", i, ref, in[i]);
 }
 
  // Release the heap memory after we are done using it
  free(in);
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
