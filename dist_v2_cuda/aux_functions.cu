#include "hip/hip_runtime.h"
#include "aux_functions.h"
#include <math.h>
#include <stdio.h>

__host__ float scale(int i, int n)
{
  return ((float)i) / (n - 1);
}

__device__ float distance(float x1, float x2)
{
  return sqrt((x2 - x1)*(x2 - x1));
}

__global__ void distanceArray(float *d_out, float *d_in, float ref, int n)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  d_out[i] = distance(d_in[i], ref);
//  printf("i = %2d: dist from %f to %f is %f.\n", i, ref, d_in[i], d_out[i]);
}
