#include "hip/hip_runtime.h"
#define N 64 // Specify a constant value for array length.
#define TPB 32
#include <stdio.h>

// A scaling function to convert integers 0,1,...,N-1
// to evenly spaced floats ranging from 0 to 1.
__device__ float scale(int i, int n)
{
  return ((float)i) / (n - 1);
}

// Compute the distance between 2 points on a line.
__device__ float distance(float x1, float x2)
{
  return sqrt((x2 - x1)*(x2 - x1));
}


__global__ void distanceKernel(float *d_out, float ref, int len){
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  float x = scale(i, N);
  d_out[i] = distance(x, ref);
#  printf(“i = %2d: dist from %f to %f is %f.\n”, i, ref, x, d_out[i]);
}


int main()
{
  // Create an array of N floats (initialized to 0.0).
  // We will overwrite these values to store our results.
  float *d_out;
  hipMalloc(&d_out, N*sizeof(float));

  // Choose a reference value from which distances are measured.
  const float ref = 0.5f;

  /* for loop to scale the index to obtain coordinate value,
   * compute the distance from the reference point,
   * and store the result in the corresponding entry in out. */
  distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);
  hipFree(d_out);
  return 0;

}
